#include "hip/hip_runtime.h"
#include "GPUFilter.h"

#include <iostream>
#include <vector>
#include "settings.h"
#include "check_error.h"
#include "data.h"
#include "kernels.h"


namespace GPUFilter {

namespace {

    /**
     * @brief Recursively prescans and fills the pyramid of indices
     * @param g_index_pyramid
     * @param level_sizes
     * @param level Current level that we are working on
     * @param level_size Size (length) of the array of elements in the current level
     */
    void determineIndicesRecursive (std::vector<int*> &g_index_pyramid, std::vector<int> &level_sizes,
                                    int level, int level_size)
    {
        int shared_mem_size = 2 * THREADS_PER_BLOCK;
        int num_blocks = std::ceil(double(level_size) / shared_mem_size);

        // Allocate memory for the block sums
        int* g_block_sums_out; hipMalloc((void**)&g_block_sums_out, num_blocks*sizeof(int));

        // Here we call only prescan (without filter) because we need only to compute the sums
        // The values in the current level of the pyramid get replaced by the partial sums
        onlyPrescan<<< num_blocks, THREADS_PER_BLOCK, shared_mem_size >>>(g_index_pyramid[level],
                                                                          level_size,
                                                                          g_index_pyramid[level],
                                                                          g_block_sums_out);

        // Add the block sum values to the pyramid as a new level
        g_index_pyramid.push_back(g_block_sums_out);
        level_sizes.push_back(num_blocks);

        // If there is only one block we are in the top of the pyramid
        if (num_blocks > 1)
        {
            // Call the recursive function
            determineIndicesRecursive(g_index_pyramid, level_sizes, level+1, num_blocks);
        }
    }


    /**
     * @brief Fills the 0 and 1st levels of the index pyramid
     * These levels need to be done separately because we have to call the filter() function on each
     * input element
     * @param g_da_in Input data array
     * @param length Length of the data array
     * @param g_indices_out
     * @param g_index_pyramid_out
     * @param level_sizes_out
     */
    void firstPyramidLevel (Data* g_da_in, int length, int *g_indices_out,
                            std::vector<int*> &g_index_pyramid_out, std::vector<int> &level_sizes_out)
    {
        // Each block can process double the amount of data than the number of threads in it
        int shared_mem_size = 2 * THREADS_PER_BLOCK;
        int num_blocks = std::ceil(double(length) / shared_mem_size);

        // Array for block sums of the first level
        int* g_block_sums_out;
        hipMalloc((void**)&g_block_sums_out, num_blocks*sizeof(int));

        // We need to first call kernel with filter function, which filters the elements of the input
        // array - marks the ones we want to keep. Then, prescan on the 0/1 membership array determines
        // indices within each block
        filterPrescan<<< num_blocks, THREADS_PER_BLOCK, shared_mem_size >>>(g_da_in, length,
                                                                            g_indices_out,
                                                                            g_block_sums_out);

        // Store the results in the pyramid
        // Level 0 is of size da.size and contains partial indices of the of the filtered output
        g_index_pyramid_out.push_back(g_indices_out); level_sizes_out.push_back(length);
        // Level 1 contains numbers of filtered elements in each block from the level 0 computation
        g_index_pyramid_out.push_back(g_block_sums_out); level_sizes_out.push_back(num_blocks);
    }


    /**
     * @brief Computes indices of the filtered elements (using prescan)
     * @param g_da_in Input array of Data structures
     * @param length Length of the data array
     * @param g_indices_out Output, array of size da.size with filtered element indices
     * @return Total number of filtered elements
     */
    int determineIndices (Data* g_da_in, int length, int *g_indices_out)
    {
        // The indices will be determined by prescan. The prescan must be parallel on the GPU and recursive
        // on CPU - if the output of the prescan cannot fit in one block then we have to recursively call
        // prescan on the new array
        //
        // The skeleton of this algorithm is taken from the prescan for arrays of arbitrary sizes in CUDA
        // samples: http://developer.download.nvidia.com/compute/cuda/1.1-Beta/Projects/scanLargeArray.tar.gz
        //

        // Pyramid of partial results that will be created by calling recursive prescans (kept on a GPU)
        std::vector<int*> g_index_pyramid;
        std::vector<int>  level_sizes;  // Because we want to support arrays of arbitrary sizes - store them

        // Fill the 0 and 1 pyramid level, from level 1 we can call regular prescan of the values because we
        // do not need to filter them anymore
        firstPyramidLevel(g_da_in, length, g_indices_out, g_index_pyramid, level_sizes);

        if (level_sizes.back() > 1)
        {
            // Call the recursive prescan function
            determineIndicesRecursive(g_index_pyramid, level_sizes, 1, level_sizes.back());
        }


        // The top of the pyramid contains the total number of filtered elements (the top level was not
        // processed by prescan because it has one element)
        int output_size;
        hipMemcpy(&output_size, g_index_pyramid[g_index_pyramid.size()-1], sizeof(int), hipMemcpyDeviceToHost);
        hipFree(g_index_pyramid[g_index_pyramid.size()-1]);


        // After the whole pyramid is built we need to propagate the prescan sums all the way to the bottom
        // and add them to the partial indices
        for (int l = g_index_pyramid.size()-2; l > 0; --l)
        {
            int num_blocks = std::ceil(double(level_sizes[l-1]) / THREADS_PER_BLOCK);
            propagateSum<<< num_blocks, THREADS_PER_BLOCK >>>(g_index_pyramid[l], g_index_pyramid[l-1],
                                                              level_sizes[l-1]);

            if (l != 0) hipFree(g_index_pyramid[l]);
        }

        return output_size;
    }

}


DataArray filterArray (const DataArray &da)
{
    // Copy data to GPU
    Data* g_da_in; hipMalloc((void**)&g_da_in, da.size*sizeof(Data));
    hipMemcpy(g_da_in, da.array, da.size*sizeof(Data), hipMemcpyHostToDevice);


    // Compute indices of elements in the output array - scan
    int* g_indices_out;
    hipMalloc((void**)&g_indices_out, da.size*sizeof(int));

    int output_size = determineIndices(g_da_in, da.size, g_indices_out);


    // Copy data to the output array
    Data* g_da_out; hipMalloc((void**)&g_da_out, output_size*sizeof(Data));

    int num_blocks = std::ceil(double(da.size) / THREADS_PER_BLOCK);
    copyElementsToOutput<<< num_blocks, THREADS_PER_BLOCK >>>(g_da_in, da.size, g_indices_out, g_da_out);


    DataArray da_out(output_size);
    hipMemcpy(da_out.array, g_da_out, output_size*sizeof(Data), hipMemcpyDeviceToHost);

    hipFree(g_da_in);
    hipFree(g_indices_out);
    hipFree(g_da_out);

    return da_out;
}


bool initialize ()
{
    // Find out if there is a CUDA capable device
    int device_count;
    CHECK_ERROR(hipGetDeviceCount(&device_count));

    // Get properties of the device
    hipDeviceProp_t device_properties;
    CHECK_ERROR(hipGetDeviceProperties(&device_properties, 0));

    if (device_count == 0 || (device_properties.major == 0 && device_properties.minor == 0))
    {
        // Error, we cannot initialize
        return false;
    }
    else
    {
        // Copying a dummy to the device will initialize it
        int* gpu_dummy;
        hipMalloc((void**)&gpu_dummy, sizeof(int));
        hipFree(gpu_dummy);

        std::cout << "--------------------------------------------------------------" << std::endl;
        std::cout << "Device name:           " << device_properties.name << std::endl;
        std::cout << "Compute capability:    " << device_properties.major << "." << device_properties.minor << std::endl;
        std::cout << "Total global memory:   " << device_properties.totalGlobalMem << std::endl;
        std::cout << "Multiprocessor count:  " << device_properties.multiProcessorCount << std::endl;
        std::cout << "Max threads per block: " << device_properties.maxThreadsPerBlock << std::endl;
        std::cout << "Max threads dim:       " << device_properties.maxThreadsDim[0] << std::endl;
        std::cout << "Max grid size:         " << device_properties.maxGridSize[0] << std::endl;
        std::cout << "--------------------------------------------------------------" << std::endl;

        return true;
    }
}


}

