#include "check_error.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


void HandleError(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( error ), file, line );
//        exit( EXIT_FAILURE );
    }
}

